#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>        // for strcmp
#include <chrono>
#include <hip/hip_runtime.h>

#include <mma.h>
#include <hipblas.h>
#include <omp.h>          // OpenMP for CPU parallelization
#include <immintrin.h>    // AVX intrinsics
#include <cblas.h>        // OpenBLAS for optimized CPU operations

using namespace nvcuda;
using namespace std::chrono;

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

#define CHECK_CUBLAS(call) do { \
    hipblasStatus_t stat = call; \
    if (stat != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error at %s:%d: %d\n", __FILE__, __LINE__, stat); \
        exit(1); \
    } \
} while(0)

////////////////////////////////////////////////////////////////////////////////
// CPU Single-Core GEMM (FP32) - Baseline implementation
// Compute C = A * B (naive triple loop)
////////////////////////////////////////////////////////////////////////////////
void matmul_cpu_single_core(const float* A, const float* B, float* C, int N) {
    // Simple triple loop - no optimization
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < N; ++k) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// CPU Manual Optimized GEMM (FP32) - Multi-core + SIMD + Cache blocking
// Compute C = A * B with OpenMP + AVX + blocking (manual implementation)
////////////////////////////////////////////////////////////////////////////////
void matmul_cpu_manual_optimized(const float* A, const float* B, float* C, int N) {
    const int BLOCK_SIZE = 64;  // Cache-friendly block size

    // Initialize result matrix to zero
    #pragma omp parallel for
    for (int i = 0; i < N * N; ++i) {
        C[i] = 0.0f;
    }

    // Blocked matrix multiplication with OpenMP parallelization
    #pragma omp parallel for collapse(2) schedule(dynamic)
    for (int ii = 0; ii < N; ii += BLOCK_SIZE) {
        for (int jj = 0; jj < N; jj += BLOCK_SIZE) {
            for (int kk = 0; kk < N; kk += BLOCK_SIZE) {
                // Block boundaries
                int i_end = std::min(ii + BLOCK_SIZE, N);
                int j_end = std::min(jj + BLOCK_SIZE, N);
                int k_end = std::min(kk + BLOCK_SIZE, N);

                // Inner block computation with AVX optimization
                for (int i = ii; i < i_end; ++i) {
                    for (int j = jj; j < j_end; j += 8) {  // Process 8 elements at once with AVX
                        if (j + 8 <= j_end) {
                            // AVX vectorized inner loop
                            __m256 sum_vec = _mm256_load_ps(&C[i * N + j]);

                            for (int k = kk; k < k_end; ++k) {
                                __m256 a_vec = _mm256_broadcast_ss(&A[i * N + k]);
                                __m256 b_vec = _mm256_load_ps(&B[k * N + j]);
                                sum_vec = _mm256_fmadd_ps(a_vec, b_vec, sum_vec);
                            }

                            _mm256_store_ps(&C[i * N + j], sum_vec);
                        } else {
                            // Handle remaining elements (non-vectorizable)
                            for (int j_rem = j; j_rem < j_end; ++j_rem) {
                                for (int k = kk; k < k_end; ++k) {
                                    C[i * N + j_rem] += A[i * N + k] * B[k * N + j_rem];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// CPU OpenBLAS Optimized GEMM (FP32) - Industry-standard optimized BLAS
// Compute C = A * B using cblas_sgemm (highly optimized)
////////////////////////////////////////////////////////////////////////////////
void matmul_cpu_openblas(const float* A, const float* B, float* C, int N) {
    // OpenBLAS cblas_sgemm performs: C = alpha * A * B + beta * C
    // Parameters: (order, transA, transB, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc)
    // CblasRowMajor: row-major storage
    // CblasNoTrans: no transpose
    // M=N, N=N, K=N for square matrices
    // alpha=1.0, beta=0.0 for C = A * B
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
                N, N, N,                    // M, N, K dimensions
                1.0f,                       // alpha
                A, N,                       // A matrix, leading dimension
                B, N,                       // B matrix, leading dimension
                0.0f,                       // beta
                C, N);                      // C matrix, leading dimension
}

////////////////////////////////////////////////////////////////////////////////
// Simple CUDA-Core GEMM (FP32)
// Compute C = A * B
////////////////////////////////////////////////////////////////////////////////
__global__ void matmul_cuda_core(const float* A, const float* B, float* C, int N) {
    // blockDim: (tile, tile)
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        // simple inner loop (not optimized cache blocking), but still uses many CUDA cores in parallel
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Helper kernel: convert float array -> half (__half)
////////////////////////////////////////////////////////////////////////////////
__global__ void float_to_half_kernel(const float* in, half* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = __float2half(in[idx]);
    }
}

////////////////////////////////////////////////////////////////////////////////
// WMMA Tensor Core GEMM (FP16 inputs, FP32 accumulate) using 16x16x16 tiles
// Each warp computes one 16x16 tile (typical simple mapping).
////////////////////////////////////////////////////////////////////////////////
__global__ void matmul_wmma(const half* A, const half* B, float* C, int N) {
    // gridDim: (N/16, N/16)
    // blockDim.x must be 32 (one warp per block in this simple mapping)
    int warpM = blockIdx.x; // tile row index
    int warpN = blockIdx.y; // tile col index

    // fragments
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // iterate k in steps of 16
    for (int k = 0; k < N; k += 16) {
        // pointers to tile starting positions
        const half* tileA = A + (warpM * 16) * N + k;
        const half* tileB = B + k * N + (warpN * 16);

        // load fragments (wmma handles the necessary lanes internally)
        wmma::load_matrix_sync(a_frag, tileA, N);
        wmma::load_matrix_sync(b_frag, tileB, N);
        // matrix multiply-accumulate
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    // store result tile back to C
    float* tileC = C + (warpM * 16) * N + (warpN * 16);
    wmma::store_matrix_sync(tileC, acc_frag, N, wmma::mem_row_major);
}

////////////////////////////////////////////////////////////////////////////////
// Host-side utility: random init, verification
////////////////////////////////////////////////////////////////////////////////
void init_matrix_float(float* M, int N, unsigned int seed=123) {
    srand(seed);
    for (long long i = 0; i < (long long)N * N; ++i) {
        // small values to avoid overflow in naive kernel for large N
        M[i] = (float)(rand() % 100) / 100.0f;
    }
}

double max_abs_diff(const float* A, const float* B, int N) {
    double maxd = 0.0;
    for (long long i = 0; i < (long long)N * N; ++i) {
        double d = fabs((double)A[i] - (double)B[i]);
        if (d > maxd) maxd = d;
    }
    return maxd;
}

int main(int argc, char** argv) {
    int N = 1024; // default size (must be multiple of 16)
    bool gpu_only = false; // flag for GPU-only benchmark

    // Parse command line arguments
    if (argc >= 2) N = atoi(argv[1]);
    if (argc >= 3 && strcmp(argv[2], "--gpu-only") == 0) {
        gpu_only = true;
        printf("GPU-only mode enabled (skipping CPU benchmarks)\n");
    }

    if (N % 16 != 0) {
        fprintf(stderr, "N must be multiple of 16 for WMMA tiles. Given N=%d\n", N);
        return 1;
    }

    printf("Matrix multiply benchmark: N=%d x %d\n", N, N);

    size_t bytes_f = (size_t)N * N * sizeof(float);
    size_t bytes_h = (size_t)N * N * sizeof(half);

    // host allocations (aligned for AVX)
    float* h_A = (float*)aligned_alloc(32, bytes_f);  // 32-byte aligned for AVX
    float* h_B = (float*)aligned_alloc(32, bytes_f);
    float* h_C_cpu = (float*)aligned_alloc(32, bytes_f);
    float* h_C_cpu_opt = (float*)aligned_alloc(32, bytes_f);
    float* h_C_cpu_blas = (float*)malloc(bytes_f);  // OpenBLAS result buffer
    float* h_C_cuda = (float*)malloc(bytes_f);
    float* h_C_cublas = (float*)malloc(bytes_f);
    float* h_C_cublas_tc = (float*)malloc(bytes_f);
    float* h_C_wmma = (float*)malloc(bytes_f);

    init_matrix_float(h_A, N, 123);
    init_matrix_float(h_B, N, 456);

    // device allocations
    float *d_Af = nullptr, *d_Bf = nullptr, *d_C_cuda = nullptr, *d_C_cublas = nullptr, *d_C_cublas_tc = nullptr, *d_C_wmma = nullptr;
    half *d_Ah = nullptr, *d_Bh = nullptr;

    CHECK_CUDA(hipMalloc((void**)&d_Af, bytes_f));
    CHECK_CUDA(hipMalloc((void**)&d_Bf, bytes_f));
    CHECK_CUDA(hipMalloc((void**)&d_C_cuda, bytes_f));
    CHECK_CUDA(hipMalloc((void**)&d_C_cublas, bytes_f));
    CHECK_CUDA(hipMalloc((void**)&d_C_cublas_tc, bytes_f));
    CHECK_CUDA(hipMalloc((void**)&d_C_wmma, bytes_f)); // output as float

    CHECK_CUDA(hipMalloc((void**)&d_Ah, bytes_h));
    CHECK_CUDA(hipMalloc((void**)&d_Bh, bytes_h));

    // copy host floats to device floats
    CHECK_CUDA(hipMemcpy(d_Af, h_A, bytes_f, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Bf, h_B, bytes_f, hipMemcpyHostToDevice));

    // convert device float -> half (device kernel)
    int convert_threads = 256;
    int convert_blocks = (N * N + convert_threads - 1) / convert_threads;
    float_to_half_kernel<<<convert_blocks, convert_threads>>>(d_Af, d_Ah, N * N);
    float_to_half_kernel<<<convert_blocks, convert_threads>>>(d_Bf, d_Bh, N * N);
    CHECK_CUDA(hipGetLastError());

    // Warm up
    CHECK_CUDA(hipMemset(d_C_cuda, 0, bytes_f));
    CHECK_CUDA(hipMemset(d_C_cublas, 0, bytes_f));
    CHECK_CUDA(hipMemset(d_C_cublas_tc, 0, bytes_f));
    CHECK_CUDA(hipMemset(d_C_wmma, 0, bytes_f));

    // Initialize cuBLAS
    hipblasHandle_t cublasH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));

    // Enable Tensor Core usage for cuBLAS
    CHECK_CUBLAS(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));

    // timing with events/chrono
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    int repeat = 3;

    // Increase repeat count for small matrices to improve measurement accuracy
    if (N <= 512) {
        repeat = 10;
        printf("Using %d repetitions for accurate timing of small matrices\n", repeat);
    }

    // Variables for CPU benchmark results (will be set only if not gpu_only)
    double avg_ms_cpu = 0.0, avg_ms_cpu_opt = 0.0, avg_ms_cpu_blas = 0.0;

    if (!gpu_only) {
        // --- CPU Single-Core GEMM (baseline) ---
        printf("Running CPU single-core benchmark...\n");

        // warmup
        matmul_cpu_single_core(h_A, h_B, h_C_cpu, N);

        auto cpu_start = high_resolution_clock::now();
        for (int i = 0; i < repeat; ++i) {
            matmul_cpu_single_core(h_A, h_B, h_C_cpu, N);
        }
        auto cpu_end = high_resolution_clock::now();
        auto cpu_duration = duration_cast<microseconds>(cpu_end - cpu_start);
        avg_ms_cpu = cpu_duration.count() / (double)repeat / 1000.0; // Convert to ms

        printf("CPU single-core GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cpu, repeat);

        // --- CPU Optimized GEMM (Multi-core + SIMD + Cache blocking) ---
        printf("Running CPU optimized benchmark (OpenMP + AVX)...\n");
        printf("Using %d CPU threads\n", omp_get_max_threads());

        // warmup
        matmul_cpu_manual_optimized(h_A, h_B, h_C_cpu_opt, N);

        auto cpu_opt_start = high_resolution_clock::now();
        for (int i = 0; i < repeat; ++i) {
            matmul_cpu_manual_optimized(h_A, h_B, h_C_cpu_opt, N);
        }
        auto cpu_opt_end = high_resolution_clock::now();
        auto cpu_opt_duration = duration_cast<microseconds>(cpu_opt_end - cpu_opt_start);
        avg_ms_cpu_opt = cpu_opt_duration.count() / (double)repeat / 1000.0; // Convert to ms

        printf("CPU optimized GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cpu_opt, repeat);

        // --- CPU OpenBLAS GEMM (Industry-standard optimized BLAS) ---
        printf("Running CPU OpenBLAS benchmark (industry-standard BLAS)...\n");

        // warmup
        matmul_cpu_openblas(h_A, h_B, h_C_cpu_blas, N);

        auto cpu_blas_start = high_resolution_clock::now();
        for (int i = 0; i < repeat; ++i) {
            matmul_cpu_openblas(h_A, h_B, h_C_cpu_blas, N);
        }
        auto cpu_blas_end = high_resolution_clock::now();
        auto cpu_blas_duration = duration_cast<microseconds>(cpu_blas_end - cpu_blas_start);
        avg_ms_cpu_blas = cpu_blas_duration.count() / (double)repeat / 1000.0; // Convert to ms

        printf("CPU OpenBLAS GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cpu_blas, repeat);

    } else {
        printf("Skipping CPU benchmarks (GPU-only mode)\n");
    }

    // --- CUDA Core GEMM (naive) ---
    printf("Running CUDA naive benchmark...\n");
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // warmup
    matmul_cuda_core<<<grid, block>>>(d_Af, d_Bf, d_C_cuda, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < repeat; ++i) {
        matmul_cuda_core<<<grid, block>>>(d_Af, d_Bf, d_C_cuda, N);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_cuda = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_cuda, start, stop));
    double avg_ms_cuda = ms_cuda / (double)repeat;

    printf("CUDA-Core naive GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cuda, repeat);

    // --- cuBLAS optimized GEMM (FP32) ---
    printf("Running cuBLAS optimized benchmark...\n");
    // cuBLAS uses column-major format, so we compute C = B^T * A^T = (A * B)^T
    // Then we interpret the result as row-major C = A * B
    const float alpha = 1.0f, beta = 0.0f;

    // warmup
    CHECK_CUBLAS(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            N, N, N,
                            &alpha,
                            d_Bf, N,  // B matrix
                            d_Af, N,  // A matrix
                            &beta,
                            d_C_cublas, N));
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < repeat; ++i) {
        CHECK_CUBLAS(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                N, N, N,
                                &alpha,
                                d_Bf, N,  // B matrix
                                d_Af, N,  // A matrix
                                &beta,
                                d_C_cublas, N));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_cublas = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_cublas, start, stop));
    double avg_ms_cublas = ms_cublas / (double)repeat;

    printf("cuBLAS optimized GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cublas, repeat);

    // --- cuBLAS + Tensor Core GEMM (FP16 with automatic mixed precision) ---
    printf("Running cuBLAS + Tensor Core benchmark...\n");
    const float alpha_f = 1.0f, beta_f = 0.0f;

    // warmup
    CHECK_CUBLAS(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_f,
                             d_Bh, HIP_R_16F, N,  // B matrix (FP16)
                             d_Ah, HIP_R_16F, N,  // A matrix (FP16)
                             &beta_f,
                             d_C_cublas_tc, HIP_R_32F, N,  // C matrix (FP32)
                             HIP_R_32F,  // Computation type
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < repeat; ++i) {
        CHECK_CUBLAS(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                 N, N, N,
                                 &alpha_f,
                                 d_Bh, HIP_R_16F, N,  // B matrix (FP16)
                                 d_Ah, HIP_R_16F, N,  // A matrix (FP16)
                                 &beta_f,
                                 d_C_cublas_tc, HIP_R_32F, N,  // C matrix (FP32)
                                 HIP_R_32F,  // Computation type
                                 CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_cublas_tc = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_cublas_tc, start, stop));
    double avg_ms_cublas_tc = ms_cublas_tc / (double)repeat;

    printf("cuBLAS + Tensor Core GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_cublas_tc, repeat);

    // --- WMMA Tensor Core GEMM ---
    printf("Running WMMA manual implementation benchmark...\n");
    // grid: N/16 x N/16 (one warp per tile approach)
    dim3 grid_wmma(N / 16, N / 16);
    dim3 block_wmma(32, 1, 1); // one warp per block (simple mapping)

    // warmup
    matmul_wmma<<<grid_wmma, block_wmma>>>(d_Ah, d_Bh, d_C_wmma, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < repeat; ++i) {
        matmul_wmma<<<grid_wmma, block_wmma>>>(d_Ah, d_Bh, d_C_wmma, N);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_wmma = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_wmma, start, stop));
    double avg_ms_wmma = ms_wmma / (double)repeat;

    printf("WMMA Tensor-Core GEMM avg time: %f ms (avg over %d runs)\n", avg_ms_wmma, repeat);

    // copy back results
    CHECK_CUDA(hipMemcpy(h_C_cuda, d_C_cuda, bytes_f, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_C_cublas, d_C_cublas, bytes_f, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_C_cublas_tc, d_C_cublas_tc, bytes_f, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_C_wmma, d_C_wmma, bytes_f, hipMemcpyDeviceToHost));

    // verify differences (using CPU single-core as reference - only if CPU benchmarks were run)
    double maxd_cpu_cpu_opt = 0.0, maxd_cpu_cpu_blas = 0.0, maxd_cpu_cuda = 0.0, maxd_cpu_cublas = 0.0, maxd_cpu_cublas_tc = 0.0, maxd_cpu_wmma = 0.0;

    if (!gpu_only) {
        maxd_cpu_cpu_opt = max_abs_diff(h_C_cpu, h_C_cpu_opt, N);
        maxd_cpu_cpu_blas = max_abs_diff(h_C_cpu, h_C_cpu_blas, N);
        maxd_cpu_cuda = max_abs_diff(h_C_cpu, h_C_cuda, N);
        maxd_cpu_cublas = max_abs_diff(h_C_cpu, h_C_cublas, N);
        maxd_cpu_cublas_tc = max_abs_diff(h_C_cpu, h_C_cublas_tc, N);
        maxd_cpu_wmma = max_abs_diff(h_C_cpu, h_C_wmma, N);
    }

    printf("\n=== Performance Summary ===\n");
    if (!gpu_only) {
        printf("1. CPU single-core     : %8.3f ms (baseline)\n", avg_ms_cpu);
        printf("2. CPU manual optimized: %8.3f ms (%.2fx faster than single-core)\n",
               avg_ms_cpu_opt, avg_ms_cpu / avg_ms_cpu_opt);
        printf("3. CPU OpenBLAS        : %8.3f ms (%.2fx faster than single-core, %.2fx vs manual)\n",
               avg_ms_cpu_blas, avg_ms_cpu / avg_ms_cpu_blas, avg_ms_cpu_opt / avg_ms_cpu_blas);
        printf("4. CUDA-Core naive     : %8.3f ms (%.2fx faster than CPU single-core)\n",
               avg_ms_cuda, avg_ms_cpu / avg_ms_cuda);
        printf("5. cuBLAS optimized    : %8.3f ms (%.2fx faster than CPU single-core, %.2fx faster than CUDA naive)\n",
               avg_ms_cublas, avg_ms_cpu / avg_ms_cublas, avg_ms_cuda / avg_ms_cublas);
        printf("6. cuBLAS + TensorCore : %8.3f ms (%.2fx faster than CPU single-core, %.2fx faster than cuBLAS)\n",
               avg_ms_cublas_tc, avg_ms_cpu / avg_ms_cublas_tc, avg_ms_cublas / avg_ms_cublas_tc);
        printf("7. WMMA manual impl    : %8.3f ms (%.2fx faster than CPU single-core, %.2fx vs cuBLAS)\n",
               avg_ms_wmma, avg_ms_cpu / avg_ms_wmma, avg_ms_cublas / avg_ms_wmma);

        printf("\n=== CPU vs GPU Comparison ===\n");
        printf("Best CPU (OpenBLAS) vs Best GPU (cuBLAS+TC): %.2fx GPU advantage\n",
               avg_ms_cpu_blas / avg_ms_cublas_tc);
    } else {
        printf("GPU-only benchmark results:\n");
        printf("1. CUDA-Core naive    : %8.3f ms\n", avg_ms_cuda);
        printf("2. cuBLAS optimized   : %8.3f ms (%.2fx faster than CUDA naive)\n",
               avg_ms_cublas, avg_ms_cuda / avg_ms_cublas);
        printf("3. cuBLAS + TensorCore: %8.3f ms (%.2fx faster than cuBLAS)\n",
               avg_ms_cublas_tc, avg_ms_cublas / avg_ms_cublas_tc);
        printf("4. WMMA manual impl   : %8.3f ms (%.2fx vs cuBLAS)\n",
               avg_ms_wmma, avg_ms_cublas / avg_ms_wmma);

        printf("\n=== GPU Performance Comparison ===\n");
        printf("Best GPU (cuBLAS+TC) vs CUDA naive: %.2fx improvement\n",
               avg_ms_cuda / avg_ms_cublas_tc);
    }

    if (!gpu_only) {
        printf("\n=== Accuracy Verification (vs CPU single-core baseline) ===\n");
        printf("Max diff (CPU single vs CPU manual opt): %e\n", maxd_cpu_cpu_opt);
        printf("Max diff (CPU single vs CPU OpenBLAS)  : %e\n", maxd_cpu_cpu_blas);
        printf("Max diff (CPU single vs CUDA naive)    : %e\n", maxd_cpu_cuda);
        printf("Max diff (CPU single vs cuBLAS)        : %e\n", maxd_cpu_cublas);
        printf("Max diff (CPU single vs cuBLAS+TC)     : %e\n", maxd_cpu_cublas_tc);
        printf("Max diff (CPU single vs WMMA)          : %e\n", maxd_cpu_wmma);
    } else {
        printf("\n=== Accuracy Verification (GPU-only mode) ===\n");
        printf("GPU accuracy verification skipped (no CPU baseline in GPU-only mode)\n");
        printf("Use cuBLAS as reference for GPU implementations\n");
    }

    // Clean up cuBLAS
    CHECK_CUBLAS(hipblasDestroy(cublasH));

    // Clean up
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_Af)); CHECK_CUDA(hipFree(d_Bf));
    CHECK_CUDA(hipFree(d_C_cuda)); CHECK_CUDA(hipFree(d_C_cublas)); CHECK_CUDA(hipFree(d_C_cublas_tc)); CHECK_CUDA(hipFree(d_C_wmma));
    CHECK_CUDA(hipFree(d_Ah)); CHECK_CUDA(hipFree(d_Bh));

    // Free host memory
    free(h_A); free(h_B); free(h_C_cuda); free(h_C_cublas); free(h_C_cublas_tc); free(h_C_wmma);
    free(h_C_cpu); free(h_C_cpu_opt); free(h_C_cpu_blas);

    return 0;
}
